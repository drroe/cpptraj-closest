

#include <hip/hip_runtime.h>
#include <cstdio>

// device kernel def
__global__ void Action_noImage_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms);

////////////////////////





void Action_NoImage_Center(double *SolventMols_,double *D_, double maskCenter[3],double maxD,int  NMols, int NAtoms)
{
  double *devI2Ptr;
  double *devI1Ptr;
  double *devO1Ptr;
  int t4;
  int t2;
  double Dist;
  int solventMol;
  int solventAtom;
  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));
  hipMalloc(((void **)(&devI1Ptr)),3 * sizeof(double ));
  hipMemcpy(devI1Ptr,maskCenter,3 * sizeof(double ),hipMemcpyHostToDevice);
  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);
  dim3 dimGrid0 = dim3(NMols,1);
  dim3 dimBlock0 = dim3(NAtoms,1);


  printf("About to launch kernel.\n");
  Action_noImage_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD, NMols, NAtoms);
  printf("Done with kernel CUDA \n");
  
  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
}