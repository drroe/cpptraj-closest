
#include <hip/hip_runtime.h>
#define __rose_lt(x,y) ((x)<(y)?(x):(y))
#define __rose_gt(x,y) ((x)>(y)?(x):(y))
//this is only  used for cuda-chill
//heavy simplification
#define NsolventMolecules_ 1024
#define NsolventAtoms_ 1024

struct MolDist 
{
///< Original solvent molecule number (starts from 1).
  int mol;
///< Closest distance of solvent molecule to atoms in distanceMask.
  double D;
//AtomMask mask;  ///< Original topology solvent molecule atom mask.
///< Actual solvent atom #s to loop over.
  double solventAtoms[1024][3];
}
;
//using dist for no image 
// and kernel for when we use solute molecule center
//extracting pulling out arrays out from struct 

void Action_NoImage_Center(struct MolDist SolventMols_[1024],double maskCenter[3],double maxD)
{
  double Dist;
  int solventMol;
  int solventAtom;
//Vec3 maskCenter = frmIn.VGeometricCenter( distanceMask_ );
//standard loop 
  for (solventMol = 0; solventMol < 1024; solventMol++) {
    SolventMols_[solventMol].D = maxD;
    for (solventAtom = 0; solventAtom < 1024; solventAtom++) {
//main dist2_noImage code
//double *a1 = maskCenter.Dptr(); //center of solute molecule
//double *a2 = frmIn.XYZ(*solvent_atom);
//double *a1 = maskCenter; //center of solute molecule
//double *a2 = SolventMols_[solventMol][solventAtom];  
//double x = a1[0] - a2[0];
//double y = a1[1] - a2[1];
//double z = a1[2] - a2[2];
//Dist = (x*x + y*y + z*z);
      Dist = maskCenter[0] * SolventMols_[solventMol].solventAtoms[solventAtom][0] + maskCenter[1] * SolventMols_[solventMol].solventAtoms[solventAtom][1] + maskCenter[2] * SolventMols_[solventMol].solventAtoms[solventAtom][2];
//D_[solventMol] = Dist < D_[solventMol] ?  Dist : D_[solventMol];
      if (Dist < SolventMols_[solventMol].D) 
        SolventMols_[solventMol].D = Dist;
    }
  }
}
